#include "hip/hip_runtime.h"
/*
 * Course: High Performance Computing 2023/2024
 *
 * Lecturer: Francesco Moscato	fmoscato@unisa.it
 *
 * Author:
 * Conato Christian		0622702273		c.conato@studenti.unisa.it
 *
 * Copyright (C) 2024 - All Rights Reserved
 *
 * //This file is the parallel version with OMP and CUDA of the RB Tree Search to find a random node in the tree
 *
 * This program is free software: you can redistribute it and/or modify it under the terms of
 * the GNU General Public License as published by the Free Software Foundation, either version
 * 3 of the License, or (at your option) any later version.
 *
 * This program is distributed in the hope that it will be useful, but WITHOUT ANY WARRANTY;
 * without even the implied warranty of MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.
 * See the GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License along with ContestOMP.
 * If not, see <https://www.gnu.org/licenses/gpl-3.0.html>.
 */

#include "../Headers/RB_Tree_Generator.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <string.h>
#include <omp.h>
#include <time.h>

#define CUDA_CHECK(X) {\
 hipError_t _m_cudaStat = X;\
 if(hipSuccess != _m_cudaStat) {\
    fprintf(stderr,"\nCUDA_ERROR: %s in file %s line %d\n",\
    hipGetErrorString(_m_cudaStat), __FILE__, __LINE__);\
    exit(1);\
 } }

//===========================================================RB TREE GENERATION================================================================//

// Struct used for simplify the structure of a RB Tree node
struct SimpleNode{
    int key;
    int color;
};

/**
 * @brief Count the number of nodes in the Red-Black Tree.
 *
 * @param node The current node being checked.
 * @param sentinel The sentinel node representing NULL.
 * @return The number of nodes in the subtree rooted at the given node.
 */
int countNodes(struct Node* node, struct Node*sentinel) {
    if (node == NULL || node == sentinel) {
        return 0;
    }
    return countNodes(node->left,sentinel) + 1 + countNodes(node->right,sentinel);
}

/**
 * @brief Fill an array with the information of the Red-Black Tree in in-order traversal.
 *
 * @param node The current node being processed.
 * @param array The array to be filled.
 * @param index The current index in the array.
 * @param sentinel The sentinel node representing NULL.
 */
void fillArrayInOrder(struct Node* node, struct Node** array, int* index, struct Node* sentinel) {
    if (node != NULL && node != sentinel) {
        fillArrayInOrder(node->left, array, index, sentinel);
        array[*index] = node;
        (*index)++;
        fillArrayInOrder(node->right, array, index, sentinel);
    }
}

/**
 * @brief Create and fill an array with the information of the Red-Black Tree in sorted order.
 *
 * @param tree The Red-Black Tree.
 * @param arraySize Pointer to store the size of the created array.
 * @return The sorted array of nodes.
 */
struct Node** createSortedArray(struct RBTree* tree, int* arraySize) {
    int numNodes = countNodes(tree->root,tree->nil);
    struct Node** sortedArray = (struct Node**)malloc((numNodes+1)* sizeof(struct Node*));

    int index = 0;
    fillArrayInOrder(tree->root, sortedArray, &index, tree->nil);

    *arraySize = numNodes;
    return sortedArray;
}

int getRandomNum(int min, int max) {
    return min + rand() % (max - min + 1);
}

/**
 * @brief Create a new node with the given key and color.
 *
 * @param key The key value of the new node.
 * @param color The color of the new node (RED or BLACK).
 * @return A pointer to the newly created node.
 */
struct Node* createNode(int key, int color) {
    struct Node* newNode = (struct Node*)malloc(sizeof(struct Node));
    newNode->key = key;
    newNode->color = color;
    newNode->parent = NULL;
    newNode->left = NULL;
    newNode->right = NULL;
    return newNode;
}

/**
 * @brief Initialize a new Red-Black Tree.
 *
 * @return A pointer to the newly initialized Red-Black Tree.
 */
struct RBTree* initializeRBTree() {
    struct RBTree* tree = (struct RBTree*)malloc(sizeof(struct RBTree));
    tree->nil = createNode(0, BLACK);
    tree->root = tree->nil;
    return tree;
}

/**
 * @brief Perform a left rotation on the Red-Black Tree.
 *
 * @param tree The Red-Black Tree.
 * @param x The node to be rotated.
 */
void leftRotate(struct RBTree* tree, struct Node* x) {
    struct Node* y = x->right;
    x->right = y->left;

    if (y->left != tree->nil) {
        y->left->parent = x;
    }

    y->parent = x->parent;

    if (x->parent == tree->nil) {
        tree->root = y;
    } else if (x == x->parent->left) {
        x->parent->left = y;
    } else {
        x->parent->right = y;
    }

    y->left = x;
    x->parent = y;
}

/**
 * @brief Perform a right rotation on the Red-Black Tree.
 *
 * @param tree The Red-Black Tree.
 * @param y The node to be rotated.
 */
void rightRotate(struct RBTree* tree, struct Node* y) {
    struct Node* x = y->left;
    y->left = x->right;

    if (x->right != tree->nil) {
        x->right->parent = y;
    }

    x->parent = y->parent;

    if (y->parent == tree->nil) {
        tree->root = x;
    } else if (y == y->parent->left) {
        y->parent->left = x;
    } else {
        y->parent->right = x;
    }

    x->right = y;
    y->parent = x;
}

/**
 * @brief Fix the Red-Black Tree properties after node insertion.
 *
 * @param tree The Red-Black Tree.
 * @param z The newly inserted node.
 */
void RBInsertFixup(struct RBTree* tree, struct Node* z) {
        while (z->parent->color == RED) {
        if (z->parent == z->parent->parent->left) {
            struct Node* y = z->parent->parent->right;

            if (y->color == RED) {
                z->parent->color = BLACK;
                y->color = BLACK;
                z->parent->parent->color = RED;
                z = z->parent->parent;
            } else {
                if (z == z->parent->right) {
                    z = z->parent;
                    leftRotate(tree, z);
                }

                z->parent->color = BLACK;
                z->parent->parent->color = RED;
                rightRotate(tree, z->parent->parent);
            }
        } else {
            struct Node* y = z->parent->parent->left;

            if (y->color == RED) {
                z->parent->color = BLACK;
                y->color = BLACK;
                z->parent->parent->color = RED;
                z = z->parent->parent;
            } else {
                if (z == z->parent->left) {
                    z = z->parent;
                    rightRotate(tree, z);
                }

                z->parent->color = BLACK;
                z->parent->parent->color = RED;
                leftRotate(tree, z->parent->parent);
            }
        }
    }

    tree->root->color = BLACK;
}

/**
 * @brief Insert a new key into the Red-Black Tree.
 *
 * @param tree The Red-Black Tree.
 * @param key The key value to be inserted.
 */
void RBInsert(struct RBTree* tree, int key) {
    struct Node* z = createNode(key, RED);
    struct Node* y = tree->nil;
    struct Node* x = tree->root;

    while (x != tree->nil) {
        y = x;
        if (z->key < x->key) {
            x = x->left;
        } else {
            x = x->right;
        }
    }

    z->parent = y;

    if (y == tree->nil) {
        tree->root = z;
    } else if (z->key < y->key) {
        y->left = z;
    } else {
        y->right = z;
    }

    z->left = tree->nil;
    z->right = tree->nil;
    z->color = RED;

    RBInsertFixup(tree, z);
}

/**
 * @brief Generate a random number within the specified range [min, max].
 *
 * @param min The minimum value of the range.
 * @param max The maximum value of the range.
 * @return The generated random number.
 */
int getRandomNumber(int min, int max) {
    return min + rand() % (max - min + 1);
}

/**
 * @brief Generate and insert random elements into the Red-Black Tree.
 * The numbers generated are extracted randomly from the range (1,numElements)
 *
 * @param numElements The number of random elements to generate.
 * @param tree The Red-Black Tree to insert elements into.
 */
void randomElementGenerator(int numElements, struct RBTree* tree){
    for (int i = 0; i < numElements; ++i) {
        int randomNumber = getRandomNumber(1, numElements);
        RBInsert(tree, randomNumber);
    }
}

/**
 * @brief Generate a Red-Black Tree with random elements taken from the
 * srand() function.
 *
 * @param numElements The number of random elements to generate.
 * @param seed The seed for the random number generator.
 * @param opt Unused parameter.
 * @return A pointer to the generated Red-Black Tree.
 */
struct RBTree* generateTree(int numElements, int seed, int opt){
    srand(seed);
    struct RBTree* tree = initializeRBTree();
    randomElementGenerator(numElements,tree);
    //inOrderTraversal(tree->root);
    return tree;
}

//==============================================================OMP+CUDA========================================================================//

/**
 * @brief Prints information about the execution of the program to a CSV file.
 *
 * @param n_nodes Number of nodes in the Red-Black Tree.
 * @param opt Optimization option.
 * @param numThreads Number of OpenMP threads.
 * @param RB_creation_time Time taken for Red-Black Tree creation.
 * @param kernel_execution_time Time taken for the CUDA kernel execution.
 * @param execution_time Total execution time of the program.
 */
void printCUDAToCSV(int n_nodes, int opt, int numThreads, double RB_creation_time, float kernel_execution_time, double execution_time){
    FILE *fp2; 
    char path2[200];
    sprintf(path2, "Informations/OMP_CUDA/opt%d/%d.csv", opt, n_nodes); 
    char *filename2 = path2;
    fp2 = fopen(filename2, "a+");
    if (fp2 == NULL) {
        perror("Errore durante l'apertura del file");
        fprintf(stderr, "Impossibile aprire il file: %s\n", filename2);
    }
    #ifdef L1_CACHE
        fprintf(fp2, "OMP+CUDA_L1;%d;1024;%06f;%06f;%06f;\n", numThreads, RB_creation_time, kernel_execution_time, execution_time);
    #else
        fprintf(fp2, "OMP+CUDA;%d;1024;%06f;%06f;%06f;\n", numThreads, RB_creation_time, kernel_execution_time, execution_time);
    #endif
    fclose(fp2);
}

/**
 * @brief Function useful to print the result of the execution to a TXT file.
 * @param n_nodes The number of nodes in the Red-Black Tree.
 * @param result The result of the RB Search
 * @param valueToSearch key to search in the tree
*/
void printFoundToTXT(int n_nodes, int finalResult, int valueToSearch){
    FILE *fp2; 
    char path2[200];
    sprintf(path2, "RB_Search_Report/nodes%d/%d.txt",n_nodes,n_nodes); 
    char *filename2 = path2;
    fp2 = fopen(filename2, "a+");
    if (fp2 == NULL) {
        perror("Errore durante l'apertura del file");
        fprintf(stderr, "Impossibile aprire il file: %s\n", filename2);
    }
    if(finalResult==0)
        fprintf(fp2, "OMP+CUDA: Key %d NOT found;\n",valueToSearch);
    else
        fprintf(fp2, "OMP+CUDA: Key %d found;\n",valueToSearch);
    fclose(fp2);
}

/**
 * @brief Performs a binary search on a sorted array.
 *
 * @param dsortedArray Pointer to the sorted array in device memory.
 * @param start Starting index for the search.
 * @param end Ending index for the search.
 * @param valueToSearch Value to be searched in the array.
 * @return 1 if the value is found, 0 otherwise.
 */
__host__ __device__ int binarySearch(struct SimpleNode* dsortedArray, int start, int end, int valueToSearch){
    while (start <= end) {
        int mid = start + (end - start) / 2;
        if (dsortedArray[mid].key == valueToSearch) return 1;
        else if (dsortedArray[mid].key < valueToSearch) start = mid + 1;
        else end = mid - 1;
    }
    return 0;
}

/**
 * @brief function to find the minimum among two numbers.
 *
 * @param a first number.
 * @param b second number.
 */
__device__ int cuda_fmin(int a,int b){
    return (a<b) ? a : b;
}

/**
 * @brief CUDA kernel function to perform parallel binary search on the GPU.
 *
 * @param dsortedArray Pointer to the sorted array in device memory.
 * @param arraySizeGPU Size of the array in GPU memory.
 * @param valueToSearch Value to be searched in the array.
 * @param dfound Pointer to the variable storing the result.
 */
__global__ void RBSearchKernel(struct SimpleNode* dsortedArray, int arraySizeGPU,int valueToSearch, int* dfound) {
    //Section for dividing the workload among the threads.
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int elemPerThread = arraySizeGPU/(blockDim.x*gridDim.x);
    int remaining = arraySizeGPU % (blockDim.x*gridDim.x);
    int start = i *elemPerThread + cuda_fmin(i,remaining); //the first i elements will have 1 extra element
    elemPerThread += (i < remaining) ? 1 : 0;
    int end = start + elemPerThread - 1;

    if(start<arraySizeGPU){
        if(binarySearch(dsortedArray,start,end,valueToSearch)){
            atomicAdd(dfound,1); //function that add 1 to dfound if the element has been found
        }
    }
}

/**
 * @brief Searches for a value on the GPU using CUDA and on the CPU using OpenMP, and measures the time taken.
 *
 * @param sortedArray Array of nodes sorted by the Red-Black Tree.
 * @param arraysize Size of the sortedArray.
 * @param numThreadsOMP Number of OpenMP threads to use for CPU search.
 * @param valueToSearch Value to be searched in the array.
 * @param numThreadCUDA Number of CUDA threads per block for GPU search.
 * @return Time taken for the GPU and CPU combined search.
 */
float searchOnDevice(struct Node** sortedArray, int arraysize, int numThreadsOMP, int valueToSearch){
    // Division of the sortedArray into two arrays, the array of the host and the array of the GPU
    int arraySizeGPU=arraysize/2, arraySizeHost=arraysize/2+arraysize%2; //Calculating the sizes of the two arrays
    struct SimpleNode* dsortedArray ; //Array of the GPU
    struct SimpleNode* tempArrayGPU; //TempArray for the GPU copying the 
    struct SimpleNode* hsortedArray; //Array of the host
    tempArrayGPU = (struct SimpleNode*)malloc(arraySizeGPU* sizeof(struct SimpleNode*));
    hsortedArray = (struct SimpleNode*)malloc(arraySizeHost*sizeof(struct SimpleNode*));

    // Filling the host sortedArray
    for (int i = 0; i < arraySizeHost; i++) {
        hsortedArray[i].key = sortedArray[i]->key;
        hsortedArray[i].color = sortedArray[i]->color;
    }

    // Filling the gpu sortedArray
    for (int i = 0; i < arraySizeGPU; i++) {
        tempArrayGPU[i].key = sortedArray[arraySizeHost + i]->key;
        tempArrayGPU[i].color = sortedArray[arraySizeHost + i]->color;
    }

    // Allocation via the hipMalloc of dsortedArray of the portion of the array for the GPU
    CUDA_CHECK(hipMalloc((void**)&dsortedArray,arraySizeGPU*sizeof(struct SimpleNode)));
    // Copy of the elements in tempArrayGPU into dsortedArray
    CUDA_CHECK(hipMemcpy(dsortedArray, tempArrayGPU, arraySizeGPU*sizeof(struct SimpleNode*),hipMemcpyHostToDevice));
    
    int* dfound;
    CUDA_CHECK(hipMalloc((void**)&dfound,sizeof(int))); //Allocation via hipMalloc of dfound that will
                                                        //contain the result of the thread that will find the result 

    /*Useful section for determining the maximum number of blocksize

    int threadsperblock = 1024;
    int blockSizeLimit = 0;
    int dynamicMemSize = 0;
    int blocksize;
    int minGridSize;
    hipOccupancyMaxPotentialBlockSize(&minGridSize,&blocksize,RBSearchKernel,dynamicMemSize,blockSizeLimit);
    printf("Block size:%d",blocksize);
    int blockspergrid = ((elemPerThread-1)/blocksize+1);*/

    int workload = arraySizeGPU/10; //Useful for dividing the workload among the threads and for 
                                    //lowering the gridSize not to exceed the limit exploiting all threads in a single block
    dim3 blockSize = 1024;
    dim3 gridSize = ((workload-1)/blockSize.x+1);

    hipEvent_t start, stop;
    CUDA_CHECK(hipEventCreate(&start));
    CUDA_CHECK(hipEventCreate(&stop));
    
    #ifdef L1_CACHE
        hipFuncSetCacheConfig(reinterpret_cast<const void*>(RBSearchKernel),hipFuncCachePreferL1); //Function to prioritize the L1 cache to 48KB
    #endif

    CUDA_CHECK(hipEventRecord(start,0));
    RBSearchKernel<<<gridSize,blockSize>>>(dsortedArray,arraySizeGPU,valueToSearch,dfound);
    CUDA_CHECK(hipEventRecord(stop,0));

    int numElemPerThreadOMP = arraySizeHost/numThreadsOMP;
    int foundCPU = 0;

    if(arraySizeHost>=numThreadsOMP){
        #pragma omp parallel for reduction(+:foundCPU)
        for (int i = 0; i < numThreadsOMP; i++) {
            int start = i * numElemPerThreadOMP;
            int end = start + numElemPerThreadOMP - 1;
            if (i == numThreadsOMP - 1) 
                end = arraySizeHost - 1;
            if (binarySearch(hsortedArray, start, end, valueToSearch)) 
                foundCPU++;
        }
    } else
        printf("Insufficienti elementi, il valore %d non potrà essere cercato!",valueToSearch);

    CUDA_CHECK(hipDeviceSynchronize());

    int foundGPU;
    CUDA_CHECK(hipMemcpy(&foundGPU,dfound,sizeof(int),hipMemcpyDeviceToHost));
    int finalResult = foundGPU + foundCPU;
   
    float elapsed;
    CUDA_CHECK(hipEventElapsedTime(&elapsed,start,stop));
    elapsed=elapsed/1000.f;
    
    CUDA_CHECK(hipEventDestroy(start));
    CUDA_CHECK(hipEventDestroy(stop));
    CUDA_CHECK(hipFree(dfound));
    CUDA_CHECK(hipFree(dsortedArray));
    free(tempArrayGPU);
    free(hsortedArray);
    printFoundToTXT(arraysize, finalResult, valueToSearch);
    return elapsed;
}

/**
 * @brief Main function to execute the Red-Black Tree search program.
 *
 * @param argc Number of command-line arguments.
 * @param argv Array of command-line arguments.
 * @return 0 on successful execution.
 */
int main(int argc, char* argv[]){

    int numElements = atoi(argv[1]); // Nodes of the tree
    int seed = atoi(argv[2]); //Seed for a random number generation
    int opt = atoi(argv[3]); //Optimization taken only for the purpose 
    int numThreadsOMP = atoi(argv[4]); //Threads OMP
    int valueToSearch = atoi(argv[5]); //key to search in the tree
    omp_set_num_threads(numThreadsOMP); //Setting the number of threads

    if(argc != 6){       
		printf("Error: rbsearch argc %d <number of nodes> <seed> <optimization> <Number of thread OMP used> <value to search>\n", argc);
		exit(0);
	}
    
    // Declaration of the variables for the times
    double program_execution, create_time;
    float kernel_execution_time;

    clock_t start_execution = clock(); // Taking the start time of the program execution
    clock_t start_creation = clock(); // Taking the start time of the creation of the tree
    struct RBTree* tree = generateTree(numElements,seed,opt); //Generation of the tree
    clock_t end_creation = clock(); // Taking the end time of the creation of the tree

    // Creation of the sorted array for containing all the nodes of the 
    // RB tree ordered in a ascending order
    int arraysize;
    struct Node** sortedArray = createSortedArray(tree, &arraysize);

    //Calling the searchOnDevice function that returns the kernel execution time
    kernel_execution_time = searchOnDevice(sortedArray,arraysize,numThreadsOMP,valueToSearch);
    free(tree);
    free(sortedArray);
    clock_t end_execution = clock(); // Taking the end time of the program execution

    //Calculation of the program execution time and the creation time
    program_execution = (double)(end_execution-start_execution)/CLOCKS_PER_SEC;
    create_time = (double)(end_creation-start_creation)/CLOCKS_PER_SEC;

    // Call the function for printing the time results
    printCUDAToCSV(numElements, opt, numThreadsOMP, create_time, kernel_execution_time, program_execution);

    return 0;
}